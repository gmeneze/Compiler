#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

/* All optimizations combined */

#define BLOCK_SIZE 16
#define GRID_SIZE 160 
#define SIZE BLOCK_SIZE*BLOCK_SIZE*GRID_SIZE*GRID_SIZE
#define BLOCK_ELEMENT_COUNT BLOCK_SIZE*BLOCK_SIZE

__constant__ int const_width;

void checkresult(float *ref, float *in, float *out, float *mul, int width){

	for(int i = 0 ; i < GRID_SIZE; i++){
		for(int j = 0; j < GRID_SIZE; j++){
			float sum = 0.0f;
			int start = j * BLOCK_SIZE * width + i * BLOCK_SIZE;
			for(int ii = 0; ii < BLOCK_SIZE; ii++){
				for(int jj = 0; jj < BLOCK_SIZE; jj++){
					sum += in[start + ii * width + jj] * mul[jj];
				}
			}
			for(int ii = 0; ii < BLOCK_SIZE; ii++){
				for(int jj = 0; jj < BLOCK_SIZE; jj++){
					if(jj % 2 == 0 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 2.0 * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else if(jj % 2 == 1 && ii % 2 == 0)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else if(jj % 2 == 1 && ii % 2 == 1)
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = (-1.0) * in[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii]/sum;
					else
						ref[(j * BLOCK_SIZE + jj) * width + i * BLOCK_SIZE + ii] = 0.0f;
				}
			}
		}
	}

	for(int i = 0; i < SIZE; i++){
		if(abs(ref[i]-out[i]) > 1.e-6){
			printf("results checking failed at %d ref %f out %f\n", i, ref[i], out[i]);
			return;
		}
	}
	printf("results checking passed!\n");
}

__global__ void norm(float *in, float *out, float *mul){
	__shared__ float sharedSum[BLOCK_ELEMENT_COUNT];
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;

	if(tx >= const_width || ty >= SIZE/const_width) return;
	int start = blockIdx.y * blockDim.y * const_width + blockIdx.x * blockDim.x;
        int index = start + threadIdx.y * const_width + threadIdx.x;

	sharedSum[threadIdx.y * BLOCK_SIZE + threadIdx.x] = in[index] * mul[threadIdx.x];

        __syncthreads();
        int i = threadIdx.y * BLOCK_SIZE + threadIdx.x;
        for(int s = BLOCK_ELEMENT_COUNT/2; s > 0; s>>=1)
        {
            if(i < s)
            {
               sharedSum[i] += sharedSum[i+s];
            }
            __syncthreads();
        }

        out[index] = ((-1 * (ty%2)) + 1 + (-2 * (tx%2)) + 1) * in[index]/sharedSum[0];
}



int main(){
	float *hA_in = (float *)malloc(SIZE * sizeof(float));
	float *hA_out = (float *)malloc(SIZE * sizeof(float));
	float *hB_in = (float *)malloc(BLOCK_SIZE * sizeof(float));
	float *ref = (float *)malloc(SIZE * sizeof(float));
	float *dA_in, *dA_out, *dB_in;
        int width = BLOCK_SIZE * GRID_SIZE;

	srand(2016);

	for(int i = 0; i < SIZE; i++){
		hA_in[i] = (float)rand()/(float)RAND_MAX;
	}
	for(int i = 0; i < BLOCK_SIZE; i++){
		hB_in[i] = (float)rand()/(float)RAND_MAX;
	}

	hipMalloc((void **)&dA_in, SIZE * sizeof(float));
	hipMalloc((void **)&dA_out, SIZE * sizeof(float));
	hipMalloc((void **)&dB_in, BLOCK_SIZE * sizeof(float));

	hipMemcpy(dA_in, hA_in, SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB_in, hB_in, BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(const_width), &width, sizeof(int));
	struct timespec start, end;
	dim3 grid(GRID_SIZE, GRID_SIZE, 1);
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &start);

	norm<<<grid, block>>>(dA_in, dA_out, dB_in);

	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &end);

	printf("kernel time %fs\n", end.tv_sec - start.tv_sec + (end.tv_nsec - start.tv_nsec)/1.e9);
	hipMemcpy(hA_out, dA_out, SIZE * sizeof(float), hipMemcpyDeviceToHost);
	checkresult(ref, hA_in, hA_out, hB_in, BLOCK_SIZE * GRID_SIZE);

}
